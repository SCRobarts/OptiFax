#include "hip/hip_runtime.h"
/* 
 *OPO_CUDA_BATCH_XTAL.cu
 *
 */

#include <hip/hip_runtime_api.h> 
#include <hip/hip_math_constants.h>
#include <hipfft/hipfft.h>
#include "OPO_BATCH_HEADER.hpp"
#include <hip/hip_runtime.h>
#include ""

/*Define block size.*/
#define threadsPerBlock	128
//#define threadsPerBlock	512

/* Define imaginary constant */
#define I make_float2(0.0,1.0)
#define c0 make_float2(0.0,0.0)
//const float2 I = make_float2(0.0,1.0);
//const float2 c0 = make_float2(0.0,0.0);

__constant__ float w;
__constant__ float d_invdt2;  
__constant__ int Np = 1; 

// complex math functions
__device__ inline
float2 conjugate(float2 arg)
{
    return make_float2(arg.x, -arg.y);
}

__device__ inline
float2 complex_exp(float arg)
{
    return make_float2(cosf(arg), sinf(arg));
}

__device__ inline
float2 complex_pow(float2 arg, int n)
{
	// return make_float2(cosf(acosf(arg.x)*n), sinf(asinf(arg.y)*n));
	return make_float2(cosf(acosf(arg.x) * n), -sinf(asinf(-arg.y)*n));
}

__device__ inline
float2 complex_add(float2 const a, float2 b)
{
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ inline
float2 complex_minus(float2 a, float2 b)
{
    return make_float2(a.x - b.x, a.y - b.y);
}

__device__ inline
float2 complex_mult(float2 ab, float2 cd)
{
    return make_float2((ab.x * cd.x) - (ab.y * cd.y), (ab.x * cd.y) + (ab.y * cd.x));
}

__device__ inline
float2 complex_scale(float2 a, float s) 
{
	return make_float2(s * a.x, s * a.y);
}


__host__ __device__ inline
float fabsf2(const float2 &a) 
{ 
	return hypot( a.x , a.y );
	//return ; 
}


__device__ inline
float2 f2absf2(const float2 &a) 
{ 
	return make_float2(fabsf2(a),0.0); 
}

__global__
//__device__ inline 
 void complex_AddVec(	float2 * const A,
             			float2 const * const B,
             		//	int const gi)
 						int const N)
{
	
    /* Calculate the global linear index, assuming a 1-d grid. */
    /*
    */
    int const gi = blockDim.x * blockIdx.x + threadIdx.x;
    if (gi < N) 
    {
	
        A[gi] = complex_add(A[gi] , B[gi]);
    }
}

 __global__ 
 void complex_ScaleVec(	float2 * const A,
             			float const b,
             			int const N,
             			//float2 const c0,
             			int const gi1,
             			int const gi2)
{
    /* Calculate the global linear index, assuming a 1-d grid. */
   	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (N * blockIdx.y);

    float pos, g, bc;
	float a = 24.0;

    if (gx < N) 
    {
    	
    	if (gx > gi1 && gx < gi2) // Remove temporal aliasing     		
    	{
    		pos = (a*2.0/N) * (fabs( (float) gx - (N/2.0) ));
        	g = coshf(pos);
    		bc = (1 - (1/(g*g))) * b;
    		A[gi] = complex_scale(A[gi] , bc);
    	}
    	else
    	{
			A[gi] = complex_scale(A[gi] , b);
    	}
    	
    }
}
// Simple all in one NLFN kernel to start with.
/* NLfn = ((A0.^2).*exp(i(wt-(bdiffw0)*z)))
 * +(2*expi(-wt+(bdiffw0)*z).*abs(A0).^2)
 */
__global__
//__device__ inline 
void NLFN_kernel(float2 * const NL, 
				 float2 const * const Ax,
				 float2 const * const Ay,
				 double const dt,
				 float const w0, 
				 double bdwz,
				 //int gi) 
				 int Npoints)
{

	float arg;
	/* Calculate the global linear index, assuming a 1-d grid. */
	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * blockIdx.y);
    if (gx < Npoints)
	/*
	for (int gi =  blockDim.x * blockIdx.x + threadIdx.x;
		gi < Npoints;
		gi += blockDim.x * gridDim.x)
	*/	
    {
    
		/* NL[gi] = ((A0[gi] * A0[gi]) * (expfi(wt[gi] - (bdiffw0 * z)))) +
    				(2 * expfi(-wt[gi] + (bdiffw0 * z)) * (abs(A0[gi]) * abs(A0[gi]) )); */

    	//float arg = ((w0 * t[gx]) - bdwz);
		//float arg = ((w0 * (dt*gx)) - bdwz); // This is inherently unshifted time... 

		if (gx > (Npoints/2))
		{
			arg = ((w0 * (dt*(gx-Npoints/2))) - bdwz);
		}
		else
		{
			arg = ((w0 * (dt*(gx+Npoints/2))) - bdwz);
		}

    	float2 AxAy		= complex_mult(Ax[gi],Ay[gi]);
    	float2 abAxAy 	= complex_mult(f2absf2(Ax[gi]),f2absf2(Ay[gi]));
    	float2 expon 	= complex_exp(arg);
    	float2 dbcnjex	= complex_add((conjugate(expon)),(conjugate(expon))); 
    	
    	NL[gi] 			= complex_add(complex_mult(AxAy, expon), complex_mult(dbcnjex,abAxAy));

    }

}

// Runge-Kutta KN Step Kernel
// K1=-h*1i*G*(NLfn-1/(2*dt)*(1i/w0)*(NLfn(ind1)-NLfn(ind2)))
__global__
//__device__ inline 
void KN_kernel (float2 const * const A0,
				float2 * const KN,
				float2 * const KN2,
				float2 * const NL,
				float const * const G,
				float const w0,
				float const h,
				int const k,
				float invdt2,
				int Npoints,
				int Nsteps,
				float2 * const AN,
				float fracstep)
{
	int in1, in2;

	/* Calculate the global linear index, assuming a 1-d grid. */
 	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gy = blockIdx.y;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * gy);
    
    int const d = 1;

	float2 iby2dtw0 = complex_mult(make_float2(invdt2/(d * w0), 0.0), I);
    float2 hiG = complex_mult(I, make_float2((-1.0 * h * G[k + (gy * Nsteps)]),0.0));

    // if cell is not a boundary
    if (gx > d - 1 && gx < Npoints - d)
    {
		in1 = gi+d;
	    in2 = gi-d;
	}   
	else
	{
	    if (gx < d)
	    {
	    in1 = gi+d;
	    //in1 = gi;	
	    in2 = Npoints - d + gi;
	    //in2 = gi;
	    }
	    if (gx > Npoints - (d + 1))
    	{
    	in1 = gi - (Npoints - d);
    	//in1 = gi;
    	in2 = gi-d;
    	//in2 = gi;
	    }
	}

	float2 dNL = complex_minus(NL[in1], NL[in2]);
	KN[gi] = complex_mult(hiG, (complex_minus(NL[gi], complex_mult(iby2dtw0, dNL))));
	 AN[gi] = complex_add(A0[gi],complex_add(complex_scale(KN[gi],fracstep),complex_scale(KN2[gi],fracstep)));
	//AN[gi] = complex_add(A0[gi],complex_scale(complex_add(KN[gi],KN2[gi]),fracstep));
}


__global__ void Dispersion_kernel (	float2 * const ApFT,
									float2 * const EFT,
									float const * const alpha_w,
									float const * const beta_op,
									int Npoints,
									int stepmod,
									int chunk,
									int Nchunks,
									float * const stepmods,
									int const k)									
{
	/* Calculate the global linear index, assuming a 1-d grid. */
    //int const gi = blockDim.x * blockIdx.x + threadIdx.x;
    int sc;
    float2 bophmod = make_float2(1.0,0.0);
	//float2 bophmod;

	// 2D grid?
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gy = blockIdx.y;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * blockIdx.y);

	if (gx == k) // Ensure this is performed by a single thread.. maybe gi for batch?
	{
		stepmods[k] = stepmod;
	}

    if (gx < Npoints)
    {
    	
    	// for (sc=0;sc<stepmod;sc++)
		// {
		// 	bophmod = complex_mult(bophmod,beta_op[gi]);
		// }
	
    	//bophmod = complex_pow(beta_op[gi],stepmod);

    	bophmod = complex_exp(-beta_op[gx] * stepmod);
		bophmod = complex_scale(bophmod, powf(alpha_w[gx],stepmod));
    	EFT[gi] = complex_mult(EFT[gi],bophmod);
/*
    	if (gi > 200 && gi < 3000)
    	{
        	EFT[gi] = complex_scale(EFT[gi] , 0.001);
    	}
*/
    	//E[chunk + (gi * Nchunks)] = EFT[gi];
        ApFT[(gy * (Nchunks-1) * Npoints) + (chunk * Npoints) + gi] = EFT[gi];
    }

}
									

__global__ void Error_kernel (	float2 * const K1,
								float2 * const K2,
								float2 * const E,
								int k,
								int stepmod,
								int Npoints,
								float const maxpe,
								float const minpe,
								bool * max_min_flag)
{
	/* Calculate the global linear index, assuming a 1-d grid. */
    //int const gi = blockDim.x * blockIdx.x + threadIdx.x;
    float errfv, pcterrfv;

  	// 2D grid
    int const gx = blockDim.x * blockIdx.x + threadIdx.x;
	int const gi = (blockDim.x * blockIdx.x) + threadIdx.x + (Npoints * blockIdx.y);
    if (gx < Npoints)
    {
    	if(fabsf2(E[gi]) > 0)
    	{
	    	//errfv 	 = 0.5 * (fabsf2(complex_minus(K2[gi],K1[gi])));
			errfv 	 = (fabsf2(complex_minus(K2[gi],K1[gi])));
			//errfv 	 =  0.01 * fabsf2(make_float2(1.0,0.0));
			//errfv 	 =  0.01;
	    	pcterrfv = 100.0 * (errfv / (fabsf2(E[gi])+1));
			//pcterrfv = 0.01;

	    	if (pcterrfv >= maxpe)
	    	{
	    		max_min_flag[0] = true;
	    	} 
	    	else
	    	{
		    	if (pcterrfv >= minpe)
		    	{
		    		max_min_flag[1] = true;
		    	} 
    		}
	 	}
    }

}


__global__ void copyKernel (float2 * Eft,
							float2 * ApFT,
							int Npoints,
							int chunk,
							int Nchunks)
{
	/* Calculate the global linear index, assuming a 1-d grid. */
    //int const gi = blockDim.x * blockIdx.x + threadIdx.x;

    //if (gi < Npoints)
	for (int gi =  blockDim.x * blockIdx.x + threadIdx.x;
		gi < Npoints;
		gi += blockDim.x * gridDim.x)	
    {
    	ApFT[chunk + (gi * Nchunks)] = Eft[gi];
    }

} 

// Host function called by MEX gateway.
 void OPO_TEST_CUDA(float2* const d_E,
					float2 * const d_A0,
					float2 * const d_A1,
					float2 * const d_NL,
					float2 * const d_K1,
					float2 * const d_K2,  
				   	float const * const d_G,
					float const * const d_alpha_w,
				   	float const w0, 
				   	double const bdiffw0,
				   	float const h,
				   	int const Nsteps,
				   	int Npoints,
					int Nbatches,
				   	double const dt,
					float const * const d_beta_op,
					float const max_pct_err,
					float const min_pct_err,
					float const sel,
					float2 * const d_Ap,
					float * const d_stepmods)
{	
	// GPU variables
	float stepmod, stepend, stepprev, k, invdt2, invSize, hmod;
	double bdwz;
	int  chunk, Nchunks, gi1, gi2;
	bool* max_min_err_exc;

	// Precompute scalars:
	invdt2 	= 1.0 / (2.0 * dt);
	invSize = 1.0 / Npoints;
	k = 0.0;
	stepprev = 4.0;
	Nchunks = Nsteps / sel;
	if (Nchunks*sel < Nsteps)
	{
		Nchunks = Nchunks + 1;
	}
	// Constants defining cut off for temporal aliasing reduction
	gi1 = 6.0*(Npoints/16);
	gi2 = 10.0*(Npoints/16);

	hipHostMalloc(&max_min_err_exc, 2*sizeof(bool));

	// CUFFT plan simple API
  	hipfftHandle plan;
  	hipfftPlan1d(&plan, Npoints, HIPFFT_C2C, Nbatches);
	//hipfftPlanMany(&plan, 1, Npoints, HIPFFT_C2C, 1);

	dim3 const blocksPerGrid(((Npoints) + threadsPerBlock - 1) / threadsPerBlock, Nbatches);
	//err_reset_kernel<<<1, 1 >>>();

	for (chunk = 0; chunk < Nchunks; chunk++)
	{
		stepmod = stepprev;

		hmod = stepmod * h;

		//for (k = 0; k < sel; k = k+stepmod)
		for (k = k; k < (chunk + 1) * sel && k < Nsteps; k = k+stepmod)
		{	
		/*	
		*/
			// Compute scalars
			bdwz = bdiffw0 * k * h;
			max_min_err_exc[0] = max_min_err_exc[1] = false;

			NLFN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_NL,d_A0,d_A0,dt,w0,bdwz,Npoints);
			//hipDeviceSynchronize();
			KN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A0,d_K1,d_K1,d_NL,d_G,w0,hmod,k,invdt2,Npoints,Nsteps,d_A1,0.5);

			bdwz = bdiffw0 * (k + stepmod) * h;// full-step co-ordinate coeff

			NLFN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_NL,d_A1,d_A1,dt,w0,bdwz,Npoints);
			//hipDeviceSynchronize();	

			KN_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A0,d_K2,d_K1,d_NL,d_G,w0,hmod,k,invdt2,Npoints,Nsteps,d_E,0.5);	
			hipDeviceSynchronize();

			//Error_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_K1, d_K2, d_E, k, stepmod, Npoints, 
			Error_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A1, d_E, d_E, k, stepmod, Npoints, 
																max_pct_err, min_pct_err, max_min_err_exc);															
			//max_min_err_exc[1] = true;
			hipDeviceSynchronize();													
			
			/* */
			
			if (max_min_err_exc[0] && stepmod > 1 && (k - stepmod) > 0)
			{
				k = k - stepmod;
				stepmod = stepmod - 1;
				hmod 	= stepmod * h;
				stepprev = stepmod; 
			}
			else 
			{
				complex_ScaleVec<<<blocksPerGrid, threadsPerBlock>>>(d_E, 1, Npoints, gi1, gi2);
	
				// Transform signal
				hipfftExecC2C(plan, (hipfftComplex *) d_E,
								(hipfftComplex *) d_E,
									HIPFFT_FORWARD);						

				//Dispersion_kernel<<<Npoints/threadsPerBlock, threadsPerBlock>>>(d_Ap+(i * Nchunks), d_E+i, d_beta_op, Npoints, stepmod, chunk, Nchunks);	
				Dispersion_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_Ap, d_E, d_alpha_w, d_beta_op, Npoints, stepmod, chunk, Nchunks, d_stepmods, k);
				//hipDeviceSynchronize();

				// Transform signal back
				hipfftExecC2C(plan, (hipfftComplex *) d_E,
								(hipfftComplex *) d_A0,
									HIPFFT_BACKWARD);

				complex_ScaleVec<<<blocksPerGrid, threadsPerBlock>>>(d_A0, invSize, Npoints, 0, 0);

				if (k + stepmod >= (chunk + 1) * sel && k < ((chunk + 1) * sel) - 1)
				{
					
					stepend = (((chunk + 1) * sel) - k) - 1; 
					hmod 	= stepend * h;
					stepprev = stepmod;
					stepmod = stepend;
				}
				else
				{
					if (!max_min_err_exc[1] && !max_min_err_exc[0] && (k + stepmod + 1) < ((chunk + 1) * sel) - 1)
					{
						//stepmod = stepmod * 2;
						stepmod = stepmod + 1;
						hmod 	= stepmod * h;
					}
					
					stepprev = stepmod;

				}
			

			} // dispersion
			
		
		} // chunk
		hipDeviceSynchronize();

	} // crystal		
	hipDeviceSynchronize();

  	// Destroy CUFFT context
	hipfftDestroy(plan);

	// Release resources
	hipHostFree(max_min_err_exc);
	//hipFree(max_min_err_exc);

}